#include "hip/hip_runtime.h"
/* Copyright (C) 2020 by NavInfo Europe B.V. The Netherlands - All rights reserved
 * Information classification: Confidential
 * This content is protected by international copyright laws.
 * Reproduction and distribution is prohibited without written permission. */
#include "gpu_image_warper.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_texture_types.h>
#include <>
#include <>
#include <texture_types.h>

// TODO(jbr) Maybe check for improved casting.

__global__ void BilinearInterpolationKernel(
    uchar3* __restrict__ out_data,
    const int out_pitch,
    const uchar3* __restrict__ in_data,
    const int in_pitch,
    const int in_width,
    const int in_height,
    const float2* __restrict__ lut_data,
    const int lut_pitch,
    const int lut_width,
    const int lut_height) {
    const int tx = threadIdx.x + blockDim.x * blockIdx.x;
    const int ty = threadIdx.y + blockDim.y * blockIdx.y;

    if ((tx < lut_width) && (ty < lut_height)) {
        const float2* lut_elem_ptr = ((float2*)((char*)lut_data + ty * lut_pitch) + tx);
        const int ix = (int)floor(lut_elem_ptr->x - 0.5f);
        const float a = lut_elem_ptr->x - 0.5f - ix;
        const int iy = (int)floor(lut_elem_ptr->y - 0.5f);
        const float b = lut_elem_ptr->y - 0.5f - iy;

        const uchar3 d00 = ((ix >= 0) && (iy >= 0) && (ix < in_width) && (iy < in_height))
                               ? *((uchar3*)((char*)in_data + iy * in_pitch) + ix)
                               : make_uchar3(0, 0, 0);
        const uchar3 d10 = ((ix + 1 >= 0) && (iy >= 0) && (ix + 1 < in_width) && (iy < in_height))
                               ? *((uchar3*)((char*)in_data + iy * in_pitch) + ix + 1)
                               : make_uchar3(0, 0, 0);
        const uchar3 d01 = ((ix >= 0) && (iy + 1 >= 0) && (ix < in_width) && (iy + 1 < in_height))
                               ? *((uchar3*)((char*)in_data + (iy + 1) * in_pitch) + ix)
                               : make_uchar3(0, 0, 0);
        const uchar3 d11 = ((ix + 1 >= 0) && (iy + 1 >= 0) && (ix + 1 < in_width) && (iy + 1 < in_height))
                               ? *((uchar3*)((char*)in_data + (iy + 1) * in_pitch) + ix + 1)
                               : make_uchar3(0, 0, 0);

        float3 tmp0, tmp1;

        tmp0.x = a * d10.x + (-d00.x * a + d00.x);
        tmp0.y = a * d10.y + (-d00.y * a + d00.y);
        tmp0.z = a * d10.z + (-d00.z * a + d00.z);

        tmp1.x = a * d11.x + (-d01.x * a + d01.x);
        tmp1.y = a * d11.y + (-d01.y * a + d01.y);
        tmp1.z = a * d11.z + (-d01.z * a + d01.z);

        uchar3* out_data_elem_ptr = ((uchar3*)((char*)out_data + ty * out_pitch) + tx);
        out_data_elem_ptr->x = b * tmp1.x + (-tmp0.x * b + tmp0.x);
        out_data_elem_ptr->y = b * tmp1.y + (-tmp0.y * b + tmp0.y);
        out_data_elem_ptr->z = b * tmp1.z + (-tmp0.z * b + tmp0.z);
    }
}

namespace nie {

namespace gpu {

GpuImageWarper::GpuImageWarper(const cv::Mat& lut) : gpu_lut_(lut.cols, lut.rows), gpu_img_out_(lut.cols, lut.rows) {
    gpuErrchk(hipMemcpy2D(
        gpu_lut_.dev_ptr,
        gpu_lut_.pitch,
        lut.ptr<void>(),
        lut.step[0],                // including padding (like pitch)
        lut.cols * lut.elemSize(),  // no-padding
        lut.rows,
        hipMemcpyHostToDevice));
}

void GpuImageWarper::Warp(const cv::Mat& img_in, cv::Mat* img_out) const {
    assert(img_in.channels() == 3);

    GpuBuffer2d<uchar3> gpu_img_in(img_in.cols, img_in.rows);
    gpuErrchk(hipMemcpy2D(
        gpu_img_in.dev_ptr,
        gpu_img_in.pitch,
        img_in.ptr<void>(),
        img_in.step[0],
        img_in.cols * img_in.elemSize(),
        img_in.rows,
        hipMemcpyHostToDevice));

    int block_size = 32;
    dim3 threads(block_size, block_size);
    dim3 blocks((gpu_lut_.width + (block_size - 1)) / block_size, (gpu_lut_.height + (block_size - 1)) / block_size);

    BilinearInterpolationKernel<<<blocks, threads>>>(
        gpu_img_out_.dev_ptr,
        gpu_img_out_.pitch,
        gpu_img_in.dev_ptr,
        gpu_img_in.pitch,
        gpu_img_in.width,
        gpu_img_in.height,
        gpu_lut_.dev_ptr,
        gpu_lut_.pitch,
        gpu_lut_.width,
        gpu_lut_.height);

    img_out->create(gpu_lut_.height, gpu_lut_.width, img_in.type());
    gpuErrchk(hipMemcpy2D(
        img_out->ptr<void>(),
        img_out->step[0],
        gpu_img_out_.dev_ptr,
        gpu_img_out_.pitch,
        gpu_img_out_.byte_width(),
        gpu_img_out_.height,
        hipMemcpyDeviceToHost));
}

}  // namespace gpu

}  // namespace nie
